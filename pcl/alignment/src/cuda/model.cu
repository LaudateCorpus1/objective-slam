#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>

#include "model.h"
#include "impl/ppf_utils.hpp"
#include "kernel.h"
#include "book.h"

Model::Model(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n){
    this->initPPFs(points, normals, n);

    // key2ppfMap: associated indices ppf indices
    this->key2ppfMap = new thrust::device_vector<unsigned int>(this->modelPPFs->size());
    thrust::sequence(key2ppfMap->begin(), key2ppfMap->end());

    // hashKeys_old: array of hashKeys
    thrust::device_vector<unsigned int> *hashKeys_old =
        new thrust::device_vector<unsigned int>(this->modelPPFs->size());

    // for each ppf, compute a 32-bit hash
    ppf_hash_kernel<<<32768/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(this->modelPPFs),
                                                     RAW_PTR(hashKeys_old),
                                                     this->modelPPFs->size());
// #ifdef DEBUG
//     {
//         using namespace std;
//         /* DEBUG */
//         fprintf(stderr, "%d, %d\n", this->modelPPFs->size(), this->modelPPFs->size()/BLOCK_SIZE);
//         /* DEBUG */

//         thrust::host_vector<float4> *ppfs = new thrust::host_vector<float4>(*(this->modelPPFs));
//         for(int i = 0; i < 20; i++){
//             cout << "PPF Number: " << i << endl;
//             cout << (*ppfs)[i].x << endl;
//             cout << (*ppfs)[i].y << endl;
//             cout << (*ppfs)[i].z << endl;
//             cout << (*ppfs)[i].w << endl;
//         }

//         thrust::host_vector<unsigned int> *hah = new thrust::host_vector<unsigned int>(*hashKeys_old);
//         cout << "hashKeys_old" << endl;
//         for(int i = 0; i < 20; i++){
//             cout << (*hah)[i] << endl;
//         }
//     }
// #endif

    thrust::sort_by_key(hashKeys_old->begin(), hashKeys_old->end(), key2ppfMap->begin());

    this->hashKeys = new thrust::device_vector<unsigned int>();
    this->ppfCount = new thrust::device_vector<unsigned int>();
    histogram_destructive(*hashKeys_old, *(this->hashKeys), *(this->ppfCount));
    delete hashKeys_old;

    // create list of beginning indices of blocks of ppfs having equal hashes
    this->firstPPFIndex = new thrust::device_vector<unsigned int>(this->hashKeys->size());

    thrust::exclusive_scan(this->ppfCount->begin(),
                           this->ppfCount->end(),
                           this->firstPPFIndex->begin());

    this->votes = NULL;
    this->voteCodes = NULL;
    this->voteCounts = NULL;

}
// TODO: Deallocate memory for things not here yet
Model::~Model(){
    delete this->ppfCount;
    delete this->firstPPFIndex;
    delete this->key2ppfMap;
    if (this->votes != NULL) delete this->votes;
    if (this->voteCodes != NULL) delete this->voteCodes;
    if (this->voteCounts != NULL) delete this->voteCounts;
}

// TODO: finish
void Model::ppf_lookup(Scene *scene){

    #ifdef DEBUG
        hipEvent_t start, stop;
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));
    #endif

    // find possible starting indices of blocks matching Model hashKeys
    thrust::device_vector<unsigned int> *sceneIndices =
        new thrust::device_vector<unsigned int>(scene->getModelPPFs()->size());
    thrust::lower_bound(this->hashKeys->begin(),
                        this->hashKeys->end(),
                        scene->getHashKeys()->begin(),
                        scene->getHashKeys()->end(),
                        sceneIndices->begin());

    // Steps 1-3
    // launch voting kernel instance for each scene reference point
    unsigned int lastIndex, lastCount;
    this->votes = new thrust::device_vector<unsigned long>(scene->getModelPPFs()->size());

    // vecs_old is an array of (soon to be) sorted translation vectors
    thrust::device_vector<float3> *vecs_old =
        new thrust::device_vector<float3>(scene->getModelPPFs()->size());

    // populates parallel arrays votes and vecs_old
    ppf_vote_kernel<<<32768,BLOCK_SIZE>>>
        (RAW_PTR(scene->getHashKeys()), RAW_PTR(sceneIndices),
         RAW_PTR(this->hashKeys), RAW_PTR(this->ppfCount),
         RAW_PTR(this->firstPPFIndex), RAW_PTR(this->key2ppfMap),
         RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
         this->n, RAW_PTR(scene->getModelPoints()),
         RAW_PTR(scene->getModelNormals()), scene->numPoints(),
         RAW_PTR(this->votes), RAW_PTR(vecs_old),
         scene->getHashKeys()->size());

#ifdef DEBUG
    {
        using namespace std;

        thrust::host_vector<float3> *hah = new thrust::host_vector<float3>(*vecs_old);
        cout << "vecs_old" << endl;
        for(int i = 0; i < 20; i++){
            if((*hah)[i].y > 0){
                cout << i << ", " << (*hah)[i] << endl;
            }
        }
    }
#endif

    thrust::sort_by_key(vecs_old->begin(), vecs_old->end(), votes->begin());
    this->vecs = new thrust::device_vector<float3>();
    this->vecCounts = new thrust::device_vector<unsigned int>();
    histogram_destructive(*vecs_old, *(this->vecs), *(this->vecCounts));
    this->firstVecIndex = new thrust::device_vector<unsigned int>(this->vecs->size());
    thrust::exclusive_scan(this->vecCounts->begin(),
                           this->vecCounts->end(),
                           this->firstVecIndex->begin());
    // votes is sorted by vecs!!


    // // populates voteCodes and voteCounts, sorts votes
    // this->accumulateVotes();

    // this->vec2VoteMap = new thrust::device_vector<unsigned int>(vecs_old->size());
    // thrust::sequence(vec2VoteMap->begin(), vec2VoteMap->end());

    // thrust::sort_by_key(vecs_old->begin(), vecs_old->end(), vec2VoteMap->begin());

    // // Step 4

    // // accumulator is an n*n_angle matrix where the ith row
    // // corresponds to the translation vector vecs[i] and the jth
    // // column corresponds to the jth angle bin. accumulator[i*n_angle + j]
    // // is the number of votes that correspond to that translation vector
    // // and angle.
    // //
    // // We need to do linear indexing since using 1d array to model 2d
    // // array. A device_vector is a host-side wrapper for device
    // // memory, so we can't create a device_vector<device_vector>. We
    // // could create a vector of device_vectors on the host, but then
    // // backing memory would be non-contiguous (only vector-wise
    // // continuous).

    // unsigned int num_bins = thrust::inner_product(vecs_old->begin(), vecs_old->end() - 1,
    //                                               vecs_old->begin() + 1,
    //                                               (unsigned int) 1,
    //                                               thrust::plus<unsigned int>(),
    //                                               thrust::not_equal_to<float3>());

    // // allocated by histogram_desctructive?
    // this->vecs = new thrust::device_vector<float3>(num_bins);
    // this->vecCounts = new thrust::device_vector<unsigned int>(num_bins);

    // histogram_destructive(*vecs_old, *(this->vecs), *(this->vecCounts));
    // delete vecs_old;

    // // create list of beginning indices of blocks of ppfs having equal hashes
    // this->firstVecIndex = new thrust::device_vector<unsigned int>(this->vecs->size());

    // thrust::exclusive_scan(this->vecCounts->begin(),
    //                        this->vecCounts->end(),
    //                        this->firstVecIndex->begin());

    // Step 5
    // Can almost represent this (and Step 4) as a reduction or transformation, but not quite.
    thrust::device_vector<unsigned int> *accumulator =
        new thrust::device_vector<unsigned int>(this->vecs->size()*N_ANGLE);

     ppf_reduce_rows_kernel<<<this->vecs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(this->vecs),
                                                                          RAW_PTR(this->vecCounts),
                                                                          RAW_PTR(this->firstVecIndex),
                                                                          RAW_PTR(this->votes),
                                                                          N_ANGLE,
                                                                          RAW_PTR(accumulator),
                                                                          this->vecs->size());

    // Steps 6, 7
    thrust::device_vector<unsigned int> *maxidx =
        new thrust::device_vector<unsigned int>(this->vecs->size());
    rowwise_max(*accumulator, this->vecs->size(), N_ANGLE, *maxidx);

    thrust::device_vector<unsigned int> *scores =
        new thrust::device_vector<unsigned int>(this->vecs->size());
    ppf_score_kernel<<<this->vecs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(accumulator),
                                                                   RAW_PTR(maxidx),
                                                                   N_ANGLE, SCORE_THRESHOLD,
                                                                   RAW_PTR(scores),
                                                                   this->vecs->size());

    // Step 8, 9
    // call trans_calc_kernel
    this->transformations = new thrust::device_vector<float>(this->vecs->size()*16);

     trans_calc_kernel<<<this->vecs->size()/BLOCK_SIZE,BLOCK_SIZE>>>
             (RAW_PTR(this->vecs), RAW_PTR(this->vecCounts),
              RAW_PTR(this->firstVecIndex), RAW_PTR(this->votes),
              RAW_PTR(maxidx), RAW_PTR(scores),
              N_ANGLE,
              RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
              this->modelPoints->size(),
              RAW_PTR(scene->getModelPoints()), RAW_PTR(scene->getModelNormals()),
              scene->getModelPoints()->size(),
              RAW_PTR(this->transformations),
              this->vecs->size());

    #ifdef DEBUG
        {
            // using namespace std;
            // for (int i=0; i<maxidx->size(); i++){
            //     std::cerr << (*maxidx)[i] << std::endl;
            // }
            // std::cerr << std::endl;
            // for (int i=0; i<accumulator->size(); i++){
            //     std::cerr << (*accumulator)[i] << std::endl;
            // }
            // std::cerr << std::endl;
            // for (int i=0; i<vecs->size(); i++){
            //     std::cerr << (*vecs)[i] << std::endl;
            // }
//            std::cerr << std::endl;
//            for (int i=0; i<votes->size(); i++){
//                std::cerr << (*votes)[i] << std::endl;
//            }
            // std::cerr << std::endl;
            // for (int i=0; i<hashKeys->size(); i++){
            //     std::cerr << (*hashKeys)[i] << std::endl;
            // }
//            std::cerr << std::endl;
//            for (int i=0; i<modelPPFs->size(); i++){
//                std::cerr << (*modelPPFs)[i] << std::endl;
//            }
        }

        // end cuda timer
        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
        fprintf(stderr, "Time to lookup model:  %3.1f ms\n", elapsedTime);
    #endif
}

void Model::accumulateVotes(){
    this->voteCodes = new thrust::device_vector<unsigned long>();
    this->voteCounts = new thrust::device_vector<unsigned int>();
    histogram(*(this->votes), *(this->voteCodes), *(this->voteCounts));
}

thrust::device_vector<float>* Model::getTransformations(){
    return this->transformations;
}
