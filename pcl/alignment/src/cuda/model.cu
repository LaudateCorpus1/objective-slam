#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>

#include "model.h"
#include "impl/ppf_utils.hpp"
#include "kernel.h"
#include "book.h"

Model::Model(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n){
    this->initPPFs(points, normals, n);

    // for each ppf, compute a 32-bit hash and concatenate it with
    // a 32-bit int representing the index of the ppf in d_ppfs
    thrust::device_vector<unsigned long> *d_codes =
        new thrust::device_vector<unsigned long>(this->modelPPFs->size());

    ppf_encode_kernel<<<this->modelPPFs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(this->modelPPFs),
                                                                         RAW_PTR(d_codes),
                                                                         this->modelPPFs->size());
    thrust::sort(d_codes->begin(), d_codes->end());

    // split codes into hashKeys_old, array of hashKeys (high 32 bits)
    // and key2ppfMap, the associated indices (low 32 bits)
    this->key2ppfMap = new thrust::device_vector<unsigned int>(this->modelPPFs->size());
    thrust::device_vector<unsigned int> *hashKeys_old =
        new thrust::device_vector<unsigned int>(this->modelPPFs->size());

    ppf_decode_kernel<<<this->modelPPFs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(d_codes),
                                                                          RAW_PTR(this->key2ppfMap),
                                                                          RAW_PTR(hashKeys_old),
                                                                          this->modelPPFs->size());
    delete d_codes;

    this->hashKeys = new thrust::device_vector<unsigned int>();
    this->ppfCount = new thrust::device_vector<unsigned int>();
    histogram_destructive(*hashKeys_old, *(this->hashKeys), *(this->ppfCount));
    delete hashKeys_old;

    // create list of beginning indices of blocks of ppfs having equal hashes
    this->firstPPFIndex = new thrust::device_vector<unsigned int>(this->hashKeys->size());

    thrust::exclusive_scan(this->ppfCount->begin(),
                           this->ppfCount->end(),
                           this->firstPPFIndex->begin());

    this->votes = NULL;
    this->voteCodes = NULL;
    this->voteCounts = NULL;
}
// TODO: Deallocate memory for things not here yet
Model::~Model(){
    delete this->ppfCount;
    delete this->firstPPFIndex;
    delete this->key2ppfMap;
    if (this->votes != NULL) delete this->votes;
    if (this->voteCodes != NULL) delete this->voteCodes;
    if (this->voteCounts != NULL) delete this->voteCounts;
}

// TODO: finish
void Model::ppf_lookup(Scene *scene){

    // find possible starting indices of blocks matching Model hashKeys
    thrust::device_vector<unsigned int> *sceneIndices =
        new thrust::device_vector<unsigned int>(scene->getModelPPFs()->size());
    thrust::lower_bound(this->hashKeys->begin(),
                        this->hashKeys->end(),
                        scene->getHashKeys()->begin(),
                        scene->getHashKeys()->end(),
                        sceneIndices->begin());

    thrust::device_vector<unsigned int> *found_ppf_starts =
        new thrust::device_vector<unsigned int>(scene->getModelPPFs()->size());
    thrust::device_vector<unsigned int> *found_ppf_count =
        new thrust::device_vector<unsigned int>(scene->getModelPPFs()->size());

    // Steps 1, 3
    // launch voting kernel instance for each scene reference point
    this->votes = new thrust::device_vector<unsigned long>(scene->getModelPPFs()->size());
    // vecCodes is an array of [trans vec|idx] packed as float4's
    thrust::device_vector<float4> *vecCodes = new thrust::device_vector<float4>(scene->getModelPPFs()->size());
    // populates parallel arrays votes and vecCodes
    ppf_vote_kernel<<<n/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(scene->getHashKeys()), RAW_PTR(sceneIndices),
                                                 RAW_PTR(this->hashKeys), RAW_PTR(this->ppfCount),
                                                 RAW_PTR(this->firstPPFIndex), RAW_PTR(this->key2ppfMap),
                                                 RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
                                                 this->n, RAW_PTR(scene->getModelPoints()),
                                                 RAW_PTR(scene->getModelNormals()), scene->numPoints(),
                                                 RAW_PTR(this->votes), RAW_PTR(vecCodes),
                                                 scene->numPoints());
    // populates voteCodes and voteCounts, sorts votes
    this->accumulateVotes();

    thrust::sort(vecCodes->begin(), vecCodes->end());

    // TODO: fix segfault that will happen here, vecs not initialized yet
    this->key2VecMap = new thrust::device_vector<unsigned int>(vecs->size());
    // vecs_old is an array of sorted translation vectors
    thrust::device_vector<float3> *vecs_old =
        new thrust::device_vector<float3>(this->modelPPFs->size());

    vec_decode_kernel<<<this->modelPPFs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(vecCodes),
                                                                         RAW_PTR(this->key2VecMap),
                                                                         RAW_PTR(vecs_old),
                                                                         this->modelPPFs->size());
    delete vecCodes;

    // Step 3, 4

    // accumulator is an n*n_angle matrix where the ith row
    // corresponds to the translation vector vecs[i] and the jth
    // column corresponds to the jth angle bin. accumulator[i*n_angle + j]
    // is the number of votes that correspond to that translation vector
    // and angle.
    //
    // We need to do linear indexing since using 1d array to model 2d
    // array. A device_vector is a host-side wrapper for device
    // memory, so we can't create a device_vector<device_vector>. We
    // could create a vector of device_vectors on the host, but then
    // backing memory would be non-contiguous (only vector-wise
    // continuous).
    thrust::device_vector<unsigned int> *accumulator =
        new thrust::device_vector<unsigned int>(this->vecs->size()*n_angle);

    unsigned int num_bins = thrust::inner_product(vecs_old->begin(), vecs_old->end() - 1,
                                                  vecs_old->begin() + 1,
                                                  (unsigned int) 1,
                                                  thrust::plus<unsigned int>(),
                                                  thrust::not_equal_to<float3>());

    this->vecs = new thrust::device_vector<float3>(num_bins);
    this->vecCounts = new thrust::device_vector<unsigned int>();

    histogram_destructive(*vecs_old, *(this->vecs), *(this->vecCounts));
    delete vecs_old;

    // create list of beginning indices of blocks of ppfs having equal hashes
    this->firstVecIndex = new thrust::device_vector<unsigned int>(this->vecs->size());

    thrust::exclusive_scan(this->vecCounts->begin(),
                           this->vecCounts->end(),
                           this->firstVecIndex->begin());

    // Step 5
    // Can almost represent this (and Step 4) as a reduction or transformation, but not quite.
    ppf_reduce_rows_kernel<<<this->vecs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(this->vecs),
                                                                         RAW_PTR(this->vecCounts),
                                                                         RAW_PTR(this->firstVecIndex),
                                                                         RAW_PTR(this->key2VecMap),
                                                                         RAW_PTR(this->voteCodes),
                                                                         RAW_PTR(this->voteCounts),
                                                                         n_angle,
                                                                         RAW_PTR(accumulator),
                                                                         this->vecs->size());

    // Steps 6, 7
    thrust::device_vector<unsigned int> *maxidx =
        new thrust::device_vector<unsigned int>(this->vecs->size());
    rowwise_max(*accumulator, this->vecs->size(), n_angle, *maxidx);

    thrust::device_vector<unsigned int> *scores =
        new thrust::device_vector<unsigned int>(this->vecs->size());
    ppf_score_kernel<<<this->vecs->size()/BLOCK_SIZE,BLOCK_SIZE>>>(RAW_PTR(accumulator),
                                                                   RAW_PTR(maxidx),
                                                                   n_angle, score_threshold,
                                                                   RAW_PTR(scores),
                                                                   this->vecs->size());

    // Step 8, 9
    // call trans calc kernel




}

void Model::accumulateVotes(){
    this->voteCodes = new thrust::device_vector<unsigned long>();
    this->voteCounts = new thrust::device_vector<unsigned int>();
    histogram(*(this->votes), *(this->voteCodes), *(this->voteCounts));
}
