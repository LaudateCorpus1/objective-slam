#include "hip/hip_runtime.h"
#include <string.h>
#include "kernel.h"

// FNV-1a hash function
// http://programmers.stackexchange.com/questions/49550/which-hashing-algorithm-is-best-for-uniqueness-and-speed
__device__ unsigned int hash(void *f, int n){
    char *s = (char *) f;
    unsigned int hash = 2166136261;
    while(n--){
        hash ^= *s++;
        hash *= 16777619;
    }
    return hash;
}

__device__ __forceinline__ float dot(float3 v1, float3 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ __forceinline__ float dot(float4 v1, float4 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z + v1.w*v2.w;
}


__device__ __forceinline__ float norm(float3 v){
    return sqrtf(dot(v, v));
}

__device__  float3 cross(float3 u, float3 v){
    float3 w = {u.y*v.z - u.z*v.y,
                u.z*v.x - u.x*v.z,
                u.x*v.y - u.y*v.z};
    return w;
}

__device__ __forceinline__ float quant_downf(float x, float y){
    return x - fmodf(x, y);
}

__device__ float4 disc_feature(float4 f, float d_dist, float d_angle){
    f.x = quant_downf(f.x, d_dist);
    f.y = quant_downf(f.y, d_angle);
    f.z = quant_downf(f.z, d_angle);
    f.w = quant_downf(f.w, d_angle);
    return f;
}

__device__ float4 compute_ppf(float3 p1, float3 n1, float3 p2, float3 n2){
    float3 d;
    d.x = p2.x - p1.x;
    d.y = p2.y - p1.y;
    d.z = p2.z - p1.z;

    float4 f;
    f.x = norm(d);
    f.y = acosf(dot(n1,d) / (norm(n1)*norm(d)));
    f.z = acosf(dot(n2,d) / (norm(n2)*norm(d)));
    f.w = acosf(dot(n1,n2) / (norm(n1)*norm(n2)));

    return f;
}

__device__ void trans(float3 v, float T[4][4]){
    memset(T, 0, sizeof(T));
    T[0][0] = 1;
    T[1][1] = 1;
    T[2][2] = 1;
    T[3][3] = 1;
    T[0][3] = v.x;
    T[1][3] = v.y;
    T[2][3] = v.z;
}

__device__ void rotx(float theta, float T[4][4]){
    memset(T, 0, sizeof(T));
    T[0][0] = 1;
    T[1][1] = cosf(theta);
    T[2][1] = sinf(theta);
    T[1][2] = -1*T[2][1];
    T[2][2] = T[1][1];
    T[3][3] = 1;
}

__device__ void roty(float theta, float T[4][4]){
    memset(T, 0, sizeof(T));
    T[0][0] = cosf(theta);
    T[0][2] = sinf(theta);
    T[1][1] = 1;
    T[2][0] = -1*T[0][2];
    T[2][2] = T[0][0];
    T[3][3] = 1;
}

__device__ void rotz(float theta, float T[4][4]){
    memset(T, 0, sizeof(T));
    T[0][0] = cosf(theta);
    T[1][0] = sinf(theta);
    T[0][1] = -1*T[1][0];
    T[1][1] = T[0][0];
    T[2][2] = 1;
    T[3][3] = 1;
}

__device__ void mat4f_mul(const float A[4][4],
                          const float B[4][4],
                          float C[4][4]){
    memset(C, 0, sizeof(C));
    for(int i = 0; i < 4; i++) {
        for(int j = 0; j < 4; j++) {
            for(int k = 0; k < 4; k++)
                C[i][j] += A[i][k]*B[k][j];
        }
    }
}

__device__ float4 mat4f_vmul(const float A[4][4], const float4 b){
    float4 *Af4 = (float4 *) A;
    float4 c;
    c.x = dot(Af4[0], b);
    c.y = dot(Af4[1], b);
    c.z = dot(Af4[2], b);
    c.w = dot(Af4[3], b);
    return c;
}

__device__ float4 homogenize(float3 v){
    float4 w = {v.x, v.y, v.z, 1};
    return w;
}

__device__ float3 dehomogenize(float4 v){
    float3 w = {v.x, v.y, v.z};
    return w;
}

__device__ float3 times(float a, float3 v){
    float3 w = {a*v.x, a*v.y, a*v.z};
    return w;
}

__device__ float4 times(float a, float4 v){
    float4 w = {a*v.x, a*v.y, a*v.z, a*v.z};
    return w;
}

__device__ float3 plus(float3 u, float3 v){
    float3 w = {u.x+v.x, u.y+v.y, u.z+v.z};
    return w;
}

__device__ float4 plus(float4 u, float4 v){
    float4 w = {u.x+v.x, u.y+v.y, u.z+v.z, u.w+v.w};
    return w;
}

__device__ float3 minus(float3 u, float3 v){
    float3 w = {u.x-v.x, u.y-v.y, u.z-v.z};
    return w;
}

__device__ float4 minus(float4 u, float4 v){
    float4 w = {u.x-v.x, u.y-v.y, u.z-v.z, u.w-v.w};
    return w;
}


__device__ void trans_model_scene(float3 m_r, float3 n_r_m, float3 m_i,
                                  float3 s_r, float3 n_r_s, float3 s_i,
                                  float T_m_g[4][4], float T_s_g[4][4], float &alpha){
    float transm[4][4], rot_y[4][4], rot_z[4][4], T_tmp[4][4];
    float4 n_tmp;

    m_r = times(-1, m_r);
    trans(m_r, transm);

    roty(atan2f(n_r_m.z, n_r_m.x), rot_y);

    n_tmp = homogenize(n_r_m);

    mat4f_vmul(rot_y, n_tmp);

    rotz(-1*atan2f(n_tmp.y, n_tmp.x), rot_z);

    mat4f_mul(rot_z, rot_y, T_tmp);
    mat4f_mul(T_tmp, transm, T_m_g);


    s_r = times(-1, s_r);
    trans(s_r, transm);

    roty(atan2f(n_r_s.z, n_r_s.x), rot_y);

    n_tmp = homogenize(n_r_s);

    mat4f_vmul(rot_y, n_tmp);

    rotz(-1*atan2f(n_tmp.y, n_tmp.x), rot_z);

    mat4f_mul(rot_z, rot_y, T_tmp);
    mat4f_mul(T_tmp, transm, T_s_g);


    n_tmp = homogenize(m_i);
    n_tmp = mat4f_vmul(T_m_g, n_tmp);
    float3 u = dehomogenize(n_tmp);

    n_tmp = homogenize(s_i);
    n_tmp = mat4f_vmul(T_s_g, n_tmp);
    float3 v = dehomogenize(n_tmp);

    u.x = 0;
    v.x = 0;

    alpha = atan2f(cross(u, v).x, dot(u, v));
}

__global__ void ppf_kernel(float3 *points, float3 *norms, float4 *out, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count) {

        __shared__ float3 Spoints[BLOCK_SIZE];
        __shared__ float3 Snorms[BLOCK_SIZE];

        float3 thisPoint = points[idx];
        float3 thisNorm  = norms[idx];

        for(int i = 0; i < count/BLOCK_SIZE; i++){

            Spoints[ind] = points[i*BLOCK_SIZE+ind];
            Snorms[ind]  = norms[i*BLOCK_SIZE+ind];
            __syncthreads();

            for(int j = 0; j < BLOCK_SIZE; j++) {
                if((idx*count + j + i*BLOCK_SIZE) % (count+1) == 0) continue;
                out[idx*count + j + i*BLOCK_SIZE] = compute_ppf(thisPoint, thisNorm, Spoints[j], Snorms[j]);
                out[idx*count + j + i*BLOCK_SIZE] = disc_feature(out[idx*count + j + i*BLOCK_SIZE],
                                                                 d_dist, d_angle);
            }
        }
    }
}

// TODO: increase thread work
__global__ void ppf_encode_kernel(float4 *ppfs, unsigned long *codes, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count){
        unsigned int hk = hash(ppfs+idx, sizeof(float4));
        codes[idx] = (((unsigned long) hk) << 32) + idx;
    }
}

// TODO: increase thread work
__global__ void ppf_decode_kernel(unsigned long *codes, unsigned int *key2ppfMap,
                                  unsigned int *hashKeys, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    unsigned long low32 = ((unsigned long) -1) >> 32;

    if(idx < count){
        // line 11 in algorithm 1, typo on their part
        key2ppfMap[idx] = (unsigned int) (codes[idx] & low32);
        hashKeys[idx] = (unsigned int) (codes[idx] >> 32);
    }
}

// TODO: increase thread work
__global__ void ppf_hash_kernel(float4 *ppfs, unsigned int *codes, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count){
        codes[idx] = hash(ppfs+idx, sizeof(float4));
    }
}

// TODO: increase thread work
__global__ void ppf_lookup_kernel(unsigned int *sceneKeys, unsigned int *sceneIndices,
                                  unsigned int *hashKeys, unsigned int *ppfCount,
                                  unsigned int *firstPPFIndex, unsigned int *key2ppfMap,
                                  float3 *modelPoints, float3 *modelNormals, int modelSize,
                                  float3 *scenePoints, float3 *sceneNormals, int sceneSize,
                                  unsigned int *votes,
                                  int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count){
        unsigned int thisSceneKey = sceneKeys[idx];
        unsigned int thisSceneIndex = sceneIndices[idx];
        float3 thisScenePoint = scenePoints[idx];
        float3 thisSceneNormal = sceneNormals[idx];
        if (thisSceneKey != hashKeys[thisSceneIndex]){
            return;
        }
        unsigned int thisPPFCount = ppfCount[thisSceneIndex];
        unsigned int thisFirstPPFIndex = firstPPFIndex[thisSceneIndex];

        unsigned int modelPPFIndex, model_r_index, model_i_index;
        float3 model_r_point, model_r_norm, model_i_point, scene_i_point;
        for(int i = 0; i < thisPPFCount; i++){
            modelPPFIndex = key2ppfMap[thisFirstPPFIndex+i];
            model_r_index = modelPPFIndex / modelSize;
            model_i_index = modelPPFIndex % modelSize;

            model_r_point = modelPoints[model_r_index];
            model_r_norm = modelNormals[model_r_index];
            model_i_point = modelPoints[model_i_index];

            // for(int j = 0; j < sceneSize; j++){
            //     scene_i_point = scenePoints[j];
            //     trans_model_scene(model_r_point, model_r_normal, model_i_point,
            //                       thisScenePoint, thisSceneNormal, scene_i_point);

            // }

        }


    }
}

// TODO: increase thread work
__global__ void ppf_vote_kernel(float4 *ppfs, unsigned long *codes, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count){

        codes[idx] = hash(ppfs+idx, sizeof(float4));
    }

}
