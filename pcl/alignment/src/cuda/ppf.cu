#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <>    // Stops underlining of threadIdx etc.
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#include "book.h"

using namespace std;

__device__ __forceinline__ float dot(float3 v1, float3 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ __forceinline__ float norm(float3 v){
    return sqrtf(dot(v, v));
}

__device__ float4 compute_ppf(float3 p1, float3 p2, float3 n1, float3 n2){
    float3 d;
    d.x = p2.x - p1.x;
    d.y = p2.y - p1.y;
    d.z = p2.z - p1.z;

    float4 f;
    f.x = norm(d);
    f.y = acosf(dot(n1,d) / (norm(n1)*norm(d)));
    f.z = acosf(dot(n2,d) / (norm(n2)*norm(d)));
    f.w = acosf(dot(n1,n2) / (norm(n1)*norm(n2)));

    return f;
}

__global__ void ppf_wrapper(float3 *points, float3 *norms, float4 *out, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count) {
        float3 thisPoint = points[idx];
        float3 thisNorm = norms[idx];

        for(int j = 0; j < count; j++) {
            // if(j == idx) continue;
            out[ind*count + j] = compute_ppf(thisPoint, points[j], thisNorm, norms[j]);
        }

        // for(int i = 0; i < count; i+= BLOCK_SIZE){
        //     Spoints[ind] = points[i+ind];
        //     __syncthreads();

        //     for(int j = 0; j < BLOCK_SIZE; j++) {
        //         if(i+j == idx) continue;

        //         out[ind*BLOCK_SIZE + j] = compute_ppf(thisPoint, Spoints[j], thisNorm, Snorms[j]);
        //     }
        // }
    }
}


float input(int fd)    // basic input structure
{
    float x;
    read(fd, &x, sizeof(float));
    return x;
}

int ply_load_main(char *point_path, char *norm_path, int N){
    int points_fin = open(point_path, O_RDONLY);    // read in points
    int norms_fin = open(norm_path, O_RDONLY);    // read in norms
    // Array of points
    float3 *points = new float3[N];
    float3 *norms = new float3[N];

    // read in data and time
    long startTime = clock();
    for(int i = 0; i < N; i++)    // loop over rows
    {    
        points[i].x = input(points_fin);    // read in an input entry
        points[i].y = input(points_fin);
        points[i].z = input(points_fin);

        norms[i].x = input(norms_fin);    // read in an input entry
        norms[i].y = input(norms_fin);
        norms[i].z = input(norms_fin);

    }
    long finishTime = clock();

    hipDeviceProp_t  prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    float3 *d_points; // GPU version
    float3 *d_norms; // GPU version
    float4 *ppfs = new float4[N*N];
    float4 *d_ppfs; // GPU version

    HANDLE_ERROR(hipMalloc(&d_points, N*sizeof(float3)));
    HANDLE_ERROR(hipMemcpy(d_points, points, N*sizeof(float3), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc(&d_norms, N*sizeof(float3)));
    HANDLE_ERROR(hipMemcpy(d_norms, norms, N*sizeof(float3), hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc(&d_ppfs, N*N*sizeof(float4)));

    // call ppf kernel
    ppf_wrapper<<<1,1024>>>(d_points, d_norms, d_ppfs, N);
    HANDLE_ERROR(hipMemcpy(ppfs, d_ppfs, N*N*sizeof(float4), hipMemcpyDeviceToHost));

    // end timer
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);

    // write out ppfs
    for(int i = 0; i < 10; i++)    // loop over rows
    {
        cout << "PPF Number: " << i << endl;
        cout << ppfs[i].x << endl;
        cout << ppfs[i].y << endl;
        cout << ppfs[i].z << endl;
        cout << ppfs[i].w << endl;
    }

    cout<<"Data Load Time"<<" "<<(finishTime - startTime)<<" ms"<<endl;

    // Deallocate ram
    delete[] points;
    delete[] norms;
    delete[] ppfs;

    hipFree(d_points);
    hipFree(d_norms);
    hipFree(d_ppfs);

    hipDeviceReset();

    // close input file
    close(points_fin);
    close(norms_fin);
    return 0;
}

// int ppf_run(Eigen::MatrixXf &points, Eigen::MatrixXf &normals){
//     float *point_data = points.data();
//     float *normal_data = normals.data();
//     int size = points.rows()*points.cols()*sizeof(float);

//     float *dev_point_data, *dev_normal_data;
//     HANDLE_ERROR(hipMalloc((void **) &dev_point_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_point_data, point_data. size));

//     HANDLE_ERROR(hipMalloc((void **) &dev_normal_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_normal_data, point_data. size));

//     hipDeviceProp_t prop;
//     HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
//     int blocks = prop.multiProcessorCount;

//     ppf_kernel<<<blocks*2,256>>>(dev_buffer, SIZE, dev_histo);

// }
