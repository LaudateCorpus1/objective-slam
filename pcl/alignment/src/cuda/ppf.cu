// #include "book.h"

#include <iostream>
#include <ctime>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
                // Stops underlining of __global__
    // Stops underlining of threadIdx etc.

using namespace std;

__device__ __forceinline__ float dot(float3 v1, float3 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ __forceinline__ float norm(float3 v){
    return sqrtf(dot(v, v));
}

__device__ float4 compute_ppf(float3 p1, float3 p2, float3 n1, float3 n2){
    float3 d;
    d.x = p2.x - p1.x;
    d.y = p2.y - p1.y;
    d.z = p2.z - p1.z;

    float4 f;
    f.x = norm(d);
    f.y = acosf(dot(n1,d) / (norm(n1)*norm(d)));
    f.z = acosf(dot(n2,d) / (norm(n2)*norm(d)));
    f.w = acosf(dot(n1,n2) / (norm(n1)*norm(n2)));

    return f;
}

#define BLOCK_SIZE 1000

__global__ void ppf_wrapper(float3 *points, float3 *norms, float4 *out, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count) {
        __shared__ float3 Spoints[BLOCK_SIZE];
        __shared__ float3 Snorms[BLOCK_SIZE];

        float3 thisPoint = points[idx];
        float3 thisNorm = norms[idx];

        for(int i = 0; i < count; i+= BLOCK_SIZE){
            Spoints[ind] = points[i+ind];
            __syncthreads();

            for(int j = 0; j < BLOCK_SIZE; j++) {
                if(i+j == idx) continue;

                out[ind*BLOCK_SIZE + j] = compute_ppf(thisPoint, Spoints[j], thisNorm, Snorms[j]);
            }
        }
    }
}


float input(ifstream &in)    // basic input structure
{
    float x;
    in >> x;
    return x;
}

int ply_load_main(char *point_path, char *norm_path, int N)
{    

    ifstream points_fin (point_path);    // read in points
    ifstream norms_fin (norm_path);    // read in norms
    // Array of points
    float3 *points = new float3[N];
    float3 *norms = new float3[N];

    // read in data and time
    long startTime = clock();
    for(int i = 0; i < N; i++)    // loop over rows
    {    
        points[i].x = input(points_fin);    // read in an input entry
        points[i].y = input(points_fin);
        points[i].z = input(points_fin);

        norms[i].x = input(norms_fin);    // read in an input entry
        norms[i].y = input(norms_fin);
        norms[i].z = input(norms_fin);

    }
    long finishTime = clock();

    float3 *d_points; // GPU version
    float3 *d_norms; // GPU version
    float4 *ppfs = new float4[N*N];
    float4 *d_ppfs; // GPU version

    hipMalloc(&d_points, N*sizeof(float3));
    hipMemcpy(d_points, points, N*sizeof(float3), hipMemcpyHostToDevice);

    hipMalloc(&d_norms, N*sizeof(float3));
    hipMemcpy(d_norms, points, N*sizeof(float3), hipMemcpyHostToDevice);

    hipMalloc(&d_ppfs, N*N*sizeof(float4));

    // call ppf kernel
    ppf_wrapper<<<1,1024>>>(d_points, d_norms, d_ppfs, N);

    hipMemcpy(ppfs, d_ppfs, N*N*sizeof(float4), hipMemcpyDeviceToHost);

    // write out ppfs
    for(int i = 0; i < 10; i++)    // loop over rows
    {
        cout << "PPF Number: " << i << endl;
        cout << ppfs[i].x << endl;
        cout << ppfs[i].y << endl;
        cout << ppfs[i].z << endl;
        cout << ppfs[i].w << endl;
    }
    
    cout<<"Data Load Time"<<" "<<(finishTime - startTime)<<" ms"<<endl;

    // Deallocate ram
    delete[] points;
    delete[] norms;
    delete[] ppfs;

    hipFree(d_points);
    hipFree(d_norms);
    hipFree(d_ppfs);

    hipDeviceReset();

    // close input file
    points_fin.close();
    norms_fin.close();
    return 0;
}

// int ppf_run(Eigen::MatrixXf &points, Eigen::MatrixXf &normals){
//     float *point_data = points.data();
//     float *normal_data = normals.data();
//     int size = points.rows()*points.cols()*sizeof(float);

//     float *dev_point_data, *dev_normal_data;
//     HANDLE_ERROR(cudaMalloc((void **) &dev_point_data, size));
//     HANDLE_ERROR(cudaMemcpy(dev_point_data, point_data. size));

//     HANDLE_ERROR(cudaMalloc((void **) &dev_normal_data, size));
//     HANDLE_ERROR(cudaMemcpy(dev_normal_data, point_data. size));

//     cudaDeviceProp prop;
//     HANDLE_ERROR(cudaGetDeviceProperties(&prop, 0));
//     int blocks = prop.multiProcessorCount;

//     ppf_kernel<<<blocks*2,256>>>(dev_buffer, SIZE, dev_histo);

// }
