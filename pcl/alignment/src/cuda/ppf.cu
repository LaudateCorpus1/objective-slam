#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <>    // Stops underlining of threadIdx etc.
#include <sys/types.h>
#include <sys/stat.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "kernel.h"
#include "book.h"
#include "SearchStructure.h"

using namespace std;

/*#include <thrust/sort.h>*/
/*#include <thrust/device_ptr.h>*/
/*#include <thrust/inner_product.h>*/
/*#include <thrust/iterator/constant_iterator.h>*/
/*#include <thrust/scan.h>*/

/*void test_histogram(char *point_path, int N){*/

    /*// file input*/
    /*FILE *points_fin;*/
    /*size_t result1;*/

    /*points_fin = fopen(point_path, "rb");*/
    /*if(points_fin==NULL){fputs ("File error: point_fin",stderr); exit (1);}*/

    /*unsigned int *points = new unsigned int[N];*/
    /*unsigned int *d_points;*/
    /*HANDLE_ERROR(hipMalloc(&d_points, N*sizeof(unsigned int)));*/
    /*if (points == NULL) {fputs ("Memory error: points",stderr); exit (2);}*/

    /*result1 = fread(points,sizeof(unsigned int),N,points_fin);*/

    /*thrust::device_ptr<unsigned int> hashKeys_old_ptr(d_points);*/
    /*HANDLE_ERROR(hipMemcpy(d_points, points, N*sizeof(unsigned int), hipMemcpyHostToDevice));*/

    /*thrust::sort(hashKeys_old_ptr, hashKeys_old_ptr+N);*/

    /*// create histogram of hash keys*/
    /*// https://code.google.com/p/thrust/source/browse/examples/histogram.cu*/
    /*unsigned int num_bins = thrust::inner_product(hashKeys_old_ptr, hashKeys_old_ptr + N - 1,*/
                                                  /*hashKeys_old_ptr + 1,*/
                                                  /*(unsigned int) 1,*/
                                                  /*thrust::plus<unsigned int>(),*/
                                                  /*thrust::not_equal_to<unsigned int>());*/

    /*[> DEBUG <]*/
    /*fprintf(stderr, "num_bins: %d\n", num_bins);*/
    /*[> DEBUG <]*/

    /*unsigned int *hashKeys, *ppfCount;*/
    /*HANDLE_ERROR(hipMalloc(&hashKeys, num_bins*sizeof(unsigned int)));*/
    /*HANDLE_ERROR(hipMalloc(&ppfCount, num_bins*sizeof(unsigned int)));*/
    /*thrust::device_ptr<unsigned int> hashKeys_ptr(hashKeys);*/
    /*thrust::device_ptr<unsigned int> ppfCount_ptr(ppfCount);*/

    /*thrust::reduce_by_key(hashKeys_old_ptr, hashKeys_old_ptr + N,*/
                          /*thrust::constant_iterator<unsigned int>(1),*/
                          /*hashKeys_ptr,*/
                          /*ppfCount_ptr);*/

    /*unsigned int A[num_bins], B[num_bins];*/
    /*[>HANDLE_ERROR(hipMemcpy(A, hashKeys, num_bins*sizeof(unsigned int), hipMemcpyDeviceToHost));<]*/
    /*HANDLE_ERROR(hipMemcpy(B, ppfCount, num_bins*sizeof(unsigned int), hipMemcpyDeviceToHost));*/

    /*for (int i=0; i<num_bins; i++){*/
        /*fprintf(stderr, "%u: %u %u\n", i, B[i], A[i]);*/
    /*}*/
/*}*/

int ply_load_main(char *point_path, char *norm_path, int N){
    // file input
    FILE *points_fin, *norms_fin;
    size_t result1, result2;

    points_fin = fopen(point_path, "rb");
    norms_fin  = fopen(norm_path, "rb");
    if(points_fin==NULL){fputs ("File error: point_fin",stderr); exit (1);}
    if(norms_fin==NULL){fputs ("File error: norms_fin",stderr); exit (1);}

    thrust::host_vector<float3> *points = new thrust::host_vector<float3>(N);
    thrust::host_vector<float3> *norms = new thrust::host_vector<float3>(N);

    if (points == NULL) {fputs ("Memory error: points",stderr); exit (2);}
    if (norms  == NULL) {fputs ("Memory error: norms",stderr); exit (2);}

    long startTime0 = clock();
    result1 = fread(RAW_PTR(points),sizeof(float3),N,points_fin);
    result2 = fread(RAW_PTR(norms),sizeof(float3),N,norms_fin);
    long finishTime0 = clock();

    cout<<"Data Load Time"<<" "<<(finishTime0 - startTime0)<<" ms"<<endl;

    if(result1 != N){fputs ("Reading error: points",stderr); exit(3);}
    if(result2 != N){fputs ("Reading error: norms",stderr); exit(3);}

    // cuda setup
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;
    /* DEBUG */
    fprintf(stderr, "blocks: %d\n", blocks);
    /* DEBUG */

    // build model description
    SearchStructure *model = new SearchStructure(points, norms, N);

    // copy ppfs back to host
    thrust::host_vector<float4> *ppfs = new thrust::host_vector<float4>(*model->getModelPPFs());

    // write out ppfs
    for(int i = 0; i < 100; i++){
        cout << "PPF Number: " << i << endl;
        cout << (*ppfs)[i].x << endl;
        cout << (*ppfs)[i].y << endl;
        cout << (*ppfs)[i].z << endl;
        cout << (*ppfs)[i].w << endl;
    }

    // Deallocate ram
    delete points;
    delete norms;
    delete ppfs;

    delete model;

    hipDeviceReset();

    // close input file
    fclose(points_fin);
    fclose(norms_fin);
    return 0;
}

// int ppf_run(Eigen::MatrixXf &points, Eigen::MatrixXf &normals){
//     float *point_data = points.data();
//     float *normal_data = normals.data();
//     int size = points.rows()*points.cols()*sizeof(float);

//     float *dev_point_data, *dev_normal_data;
//     HANDLE_ERROR(hipMalloc((void **) &dev_point_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_point_data, point_data. size));

//     HANDLE_ERROR(hipMalloc((void **) &dev_normal_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_normal_data, point_data. size));

//     hipDeviceProp_t prop;
//     HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
//     int blocks = prop.multiProcessorCount;

//     ppf_kernel<<<blocks*2,256>>>(dev_buffer, SIZE, dev_histo);

// }
