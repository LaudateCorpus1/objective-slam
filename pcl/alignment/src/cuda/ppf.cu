#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <>    // Stops underlining of threadIdx etc.
#include <sys/types.h>
#include <sys/stat.h>
#include <hip/hip_math_constants.h>
#include <functional>
#include <thrust/device_vector.h>

#include "book.h"
#include "SearchStructure.h"

#define BLOCK_SIZE 256

__const__ int n_angle = 32;
__const__ float d_angle = 2*HIP_PI_F/n_angle;
__const__ float d_dist = 0.05;

using namespace std;

__device__ __forceinline__ float dot(float3 v1, float3 v2){
    return (v1.x*v2.x + v1.y*v2.y + v1.z*v2.z);
}

__device__ __forceinline__ float norm(float3 v){
    return sqrtf(dot(v, v));
}


__device__ float4 disc_feature(float4 f, float d_dist, float d_angle){
    f.x = f.x - fmodf(f.x, d_dist);
    f.y = f.y - fmodf(f.y, d_angle);
    f.z = f.z - fmodf(f.z, d_angle);
    f.w = f.w - fmodf(f.w, d_angle);
    return f;
}

__device__ float4 compute_ppf(float3 p1, float3 n1, float3 p2, float3 n2){
    float3 d;
    d.x = p2.x - p1.x;
    d.y = p2.y - p1.y;
    d.z = p2.z - p1.z;

    float4 f;
    f.x = norm(d);
    f.y = acosf(dot(n1,d) / (norm(n1)*norm(d)));
    f.z = acosf(dot(n2,d) / (norm(n2)*norm(d)));
    f.w = acosf(dot(n1,n2) / (norm(n1)*norm(n2)));

    return f;
}

__global__ void ppf_kernel(float3 *points, float3 *norms, float4 *out, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    if(idx < count) {

        __shared__ float3 Spoints[BLOCK_SIZE];
        __shared__ float3 Snorms[BLOCK_SIZE];

        float3 thisPoint = points[idx];
        float3 thisNorm  = norms[idx];

        for(int i = 0; i < count/BLOCK_SIZE; i++){

            Spoints[ind] = points[i*BLOCK_SIZE+ind];
            Snorms[ind]  = norms[i*BLOCK_SIZE+ind];
            __syncthreads();

            for(int j = 0; j < BLOCK_SIZE; j++) {
                if((idx*count + j + i*BLOCK_SIZE) % (count+1) == 0) continue;
                out[idx*count + j + i*BLOCK_SIZE] = compute_ppf(thisPoint, thisNorm, Spoints[j], Snorms[j]);
                out[idx*count + j + i*BLOCK_SIZE] = disc_feature(out[idx*count + j + i*BLOCK_SIZE],
                                                                 d_dist, d_angle);
            }
        }
    }
}

/*#include <thrust/sort.h>*/
/*#include <thrust/device_ptr.h>*/
/*#include <thrust/inner_product.h>*/
/*#include <thrust/iterator/constant_iterator.h>*/
/*#include <thrust/scan.h>*/

/*void test_histogram(char *point_path, int N){*/

    /*// file input*/
    /*FILE *points_fin;*/
    /*size_t result1;*/

    /*points_fin = fopen(point_path, "rb");*/
    /*if(points_fin==NULL){fputs ("File error: point_fin",stderr); exit (1);}*/

    /*unsigned int *points = new unsigned int[N];*/
    /*unsigned int *d_points;*/
    /*HANDLE_ERROR(hipMalloc(&d_points, N*sizeof(unsigned int)));*/
    /*if (points == NULL) {fputs ("Memory error: points",stderr); exit (2);}*/

    /*result1 = fread(points,sizeof(unsigned int),N,points_fin);*/

    /*thrust::device_ptr<unsigned int> hashKeys_old_ptr(d_points);*/
    /*HANDLE_ERROR(hipMemcpy(d_points, points, N*sizeof(unsigned int), hipMemcpyHostToDevice));*/

    /*thrust::sort(hashKeys_old_ptr, hashKeys_old_ptr+N);*/

    /*// create histogram of hash keys*/
    /*// https://code.google.com/p/thrust/source/browse/examples/histogram.cu*/
    /*unsigned int num_bins = thrust::inner_product(hashKeys_old_ptr, hashKeys_old_ptr + N - 1,*/
                                                  /*hashKeys_old_ptr + 1,*/
                                                  /*(unsigned int) 1,*/
                                                  /*thrust::plus<unsigned int>(),*/
                                                  /*thrust::not_equal_to<unsigned int>());*/

    /*[> DEBUG <]*/
    /*fprintf(stderr, "num_bins: %d\n", num_bins);*/
    /*[> DEBUG <]*/

    /*unsigned int *hashKeys, *ppfCount;*/
    /*HANDLE_ERROR(hipMalloc(&hashKeys, num_bins*sizeof(unsigned int)));*/
    /*HANDLE_ERROR(hipMalloc(&ppfCount, num_bins*sizeof(unsigned int)));*/
    /*thrust::device_ptr<unsigned int> hashKeys_ptr(hashKeys);*/
    /*thrust::device_ptr<unsigned int> ppfCount_ptr(ppfCount);*/

    /*thrust::reduce_by_key(hashKeys_old_ptr, hashKeys_old_ptr + N,*/
                          /*thrust::constant_iterator<unsigned int>(1),*/
                          /*hashKeys_ptr,*/
                          /*ppfCount_ptr);*/

    /*unsigned int A[num_bins], B[num_bins];*/
    /*[>HANDLE_ERROR(hipMemcpy(A, hashKeys, num_bins*sizeof(unsigned int), hipMemcpyDeviceToHost));<]*/
    /*HANDLE_ERROR(hipMemcpy(B, ppfCount, num_bins*sizeof(unsigned int), hipMemcpyDeviceToHost));*/

    /*for (int i=0; i<num_bins; i++){*/
        /*fprintf(stderr, "%u: %u %u\n", i, B[i], A[i]);*/
    /*}*/
/*}*/

int ply_load_main(char *point_path, char *norm_path, int N){
    // file input
    FILE *points_fin, *norms_fin;
    size_t result1, result2;

    points_fin = fopen(point_path, "rb");
    norms_fin  = fopen(norm_path, "rb");
    if(points_fin==NULL){fputs ("File error: point_fin",stderr); exit (1);}
    if(norms_fin==NULL){fputs ("File error: norms_fin",stderr); exit (1);}

    float3 *points = new float3[N];
    float3 *norms = new float3[N];
    if (points == NULL) {fputs ("Memory error: points",stderr); exit (2);}
    if (norms  == NULL) {fputs ("Memory error: norms",stderr); exit (2);}

    long startTime0 = clock();
    result1 = fread(points,sizeof(float3),N,points_fin);
    result2 = fread(norms,sizeof(float3),N,norms_fin);
    long finishTime0 = clock();

    if(result1 != N){fputs ("Reading error: points",stderr); exit (3);}
    if(result2 != N){fputs ("Reading error: norms",stderr); exit (3);}

    /*thrust::device_vector<float3> pts(points, points + N*sizeof(float3));*/


    // cuda setup
    hipDeviceProp_t  prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;
    /* DEBUG */
    fprintf(stderr, "blocks: %d\n", blocks);
    /* DEBUG */


    // start cuda timer
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));


    // compute ppfs
    float3 *d_points, *d_norms;
    float4 *d_ppfs;
    HANDLE_ERROR(hipMalloc(&d_points, N*sizeof(float3)));
    HANDLE_ERROR(hipMalloc(&d_norms, N*sizeof(float3)));
    HANDLE_ERROR(hipMalloc(&d_ppfs, N*N*sizeof(float4)));

    HANDLE_ERROR(hipMemcpy(d_points, points, N*sizeof(float3), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_norms, norms, N*sizeof(float3), hipMemcpyHostToDevice));

    ppf_kernel<<<N/BLOCK_SIZE,BLOCK_SIZE>>>(d_points, d_norms, d_ppfs, N);


    // build model description
    SearchStructure *model = new SearchStructure(d_ppfs, N*N, BLOCK_SIZE);

    // end cuda timer
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);


    // copy ppfs back to host
    float4 *ppfs = new float4[N*N];
    HANDLE_ERROR(hipMemcpy(ppfs, d_ppfs, N*N*sizeof(float4), hipMemcpyDeviceToHost));

    // write out ppfs
    for(int i = 0; i < 100; i++){
        cout << "PPF Number: " << i << endl;
        cout << ppfs[i].x << endl;
        cout << ppfs[i].y << endl;
        cout << ppfs[i].z << endl;
        cout << ppfs[i].w << endl;
    }

    cout<<"Data Load Time"<<" "<<(finishTime0 - startTime0)<<" ms"<<endl;

    // Deallocate ram
    delete[] points;
    delete[] norms;
    delete[] ppfs;

    hipFree(d_points);
    hipFree(d_norms);
    hipFree(d_ppfs);

    delete model;

    hipDeviceReset();

    // close input file
    fclose(points_fin);
    fclose(norms_fin);
    return 0;
}

// int ppf_run(Eigen::MatrixXf &points, Eigen::MatrixXf &normals){
//     float *point_data = points.data();
//     float *normal_data = normals.data();
//     int size = points.rows()*points.cols()*sizeof(float);

//     float *dev_point_data, *dev_normal_data;
//     HANDLE_ERROR(hipMalloc((void **) &dev_point_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_point_data, point_data. size));

//     HANDLE_ERROR(hipMalloc((void **) &dev_normal_data, size));
//     HANDLE_ERROR(hipMemcpy(dev_normal_data, point_data. size));

//     hipDeviceProp_t prop;
//     HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
//     int blocks = prop.multiProcessorCount;

//     ppf_kernel<<<blocks*2,256>>>(dev_buffer, SIZE, dev_histo);

// }
